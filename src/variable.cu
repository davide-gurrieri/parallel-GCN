#include "hip/hip_runtime.h"
#include "../include/variable.cuh"

// ##################################################################################

void Variable::initialize_random()
{
    natural max_size = 0;
    for (const natural &size : sizes)
        max_size = std::max(max_size, size);

    dev_rand_states = dev_shared_ptr<RandState>(max_size);
    std::vector<RandState> states(max_size);
    for (auto &state : states)
    {
        integer x = 0, y = 0;
        while (x == 0 || y == 0)
        {
            x = rand();
            y = rand();
        }
        state.a = x;
        state.b = y;
    }
    dev_rand_states.copy_to_device(states.data());
}

// ##################################################################################

Variable::Variable(const natural size_, const bool requires_grad, const bool rand, const natural rows_, const natural cols_) : size(size_), rows(rows_), cols(cols_)
{
    dev_data = dev_shared_ptr<real>(size);
    if (requires_grad)
        dev_grad = dev_shared_ptr<real>(size);
    else
        dev_grad = dev_shared_ptr<real>();

    if (rand)
        sizes.push_back(size);
}

// ##################################################################################

__global__ void glorot_kernel(real *data, const natural size, const double scale, RandState *states)
{
    natural id = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
    for (natural i = id; i < size; i += blockDim.x * gridDim.x)
        data[i] = (unif(&states[i]) - 0.5) * scale;
}

void Variable::glorot() const
{
    if (!dev_rand_states.get())
    {
        std::cerr << "Variable::glorot: Variable must be initialized with rand = true" << std::endl;
        exit(EXIT_FAILURE);
    }
    if (rows == 0 || cols == 0)
    {
        std::cerr << "Variable::glorot: rows and cols must be set" << std::endl;
        exit(EXIT_FAILURE);
    }
    const real range = sqrtf(6.0f / (rows + cols));
    const real scale = range * 2;
    const natural n_blocks = std::min(CEIL(size, CudaParams::N_THREADS), CudaParams::N_BLOCKS);
    glorot_kernel<<<n_blocks, CudaParams::N_THREADS>>>(dev_data.get(), size, scale, dev_rand_states.get());
#ifdef DEBUG_CUDA
    CHECK_CUDA_ERROR(hipGetLastError());
#endif
}

// ##################################################################################

void Variable::zero(smart_stream stream) const
{
    dev_data.set_zero(stream);
}

void Variable::zero_grad(smart_stream stream) const
{
    dev_grad.set_zero(stream);
}

// ##################################################################################

__global__ void set_value_kernel(real *data, const real value, const natural size)
{
    natural id = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
    for (natural i = id; i < size; i += blockDim.x * gridDim.x)
        data[i] = value;
}

void Variable::set_value(const real value, smart_stream stream) const
{
    const natural n_blocks = std::min(CEIL(size, CudaParams::N_THREADS), CudaParams::N_BLOCKS);
    set_value_kernel<<<n_blocks, CudaParams::N_THREADS, 0, stream.get()>>>(dev_data.get(), value, size);
}

// ##################################################################################

void Variable::print(const std::string &what, natural col) const
{

    real *data = new real[size];
    if (what == "data")
        dev_data.copy_to_host(data);
    else if (what == "grad")
        dev_grad.copy_to_host(data);
    else
    {
        delete[] data;
        std::cerr << "Variable::print: what must be either 'data' or 'grad'" << std::endl;
        exit(EXIT_FAILURE);
    }
    int count = 0;
    for (natural i = 0; i < 20 * col && i < size; i++)
    {
        printf("%.4f ", data[i]);
        count++;
        if (count % col == 0)
            printf("\n");
    }
    delete[] data;
}

// ##################################################################################

void Variable::save(const std::string &file_name, const std::string &what, natural col) const
{

    std::vector<real> data(size);
    if (what == "data")
        dev_data.copy_to_host(data.data());
    else if (what == "grad")
        dev_grad.copy_to_host(data.data());
    else
    {
        std::cerr << "Variable::print: what must be either 'data' or 'grad'" << std::endl;
        exit(EXIT_FAILURE);
    }
    int count = 0;
    std::ofstream file(file_name);
    if (file.is_open())
    {
        for (const auto &element : data)
        {
            file << element << " ";
            count++;
            if (count % col == 0)
                file << "\n";
        }
        file.close();
        std::cout << "Vector saved to file: " << file_name << std::endl;
    }
    else
    {
        std::cerr << "Unable to open file: " << file_name << std::endl;
    }
}

// ##################################################################################